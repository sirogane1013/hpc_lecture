
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void kernel(float *a) {
  a[threadIdx.x] = threadIdx.x;
}

int main(void) {
  int N = 4;
  float *a;
  hipMallocManaged(&a, N*sizeof(float));
  kernel<<<1,N>>>(a);
  hipDeviceSynchronize();
  for (int i=0; i<N; i++)
    printf("%d %g\n",i,a[i]);
  hipFree(a);
}